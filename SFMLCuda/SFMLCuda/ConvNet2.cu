#include "hip/hip_runtime.h"
#include "ConvNet.hpp"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <cstdarg>

__global__ void ConvLayerFeed(unsigned char* input, int I_W, int I_H, int nInputs, unsigned char* output, float* kernals, int K_WH) {

	int I_Size = I_W * I_H;
	int K_WH_2 = K_WH * K_WH;

	//int nKernals = nInputs * nOutputs;
	//int kernalSize = K_WH * K_WH * nKernals;

	int kernalOffset = blockIdx.x * K_WH_2 * nInputs;

	float val = 0;
	int currKernal = 0;
	for (size_t i = 0; i < nInputs; i++)
	{
		for (size_t ky = 0; ky < K_WH; ky++)
		{
			for (size_t kx = 0; kx < K_WH; kx++)
			{
				val += input[(threadIdx.x + kx) + (threadIdx.y + ky) * I_W + I_Size * i] * kernals[kernalOffset + currKernal + ky * K_WH + kx];
			}
		}
		currKernal += K_WH_2;
	}

	if(val > 0)
		if(val < 255)
			output[blockIdx.x * blockDim.x * blockDim.y + threadIdx.x + threadIdx.y * blockDim.x] = val;
		else
			output[blockIdx.x * blockDim.x * blockDim.y + threadIdx.x + threadIdx.y * blockDim.x] = 255;
	else
		output[blockIdx.x * blockDim.x * blockDim.y + threadIdx.x + threadIdx.y * blockDim.x] = 0;
}

__global__ void MaxPoolLayer(unsigned char* input, int I_W, int I_H, unsigned char* output, int K_WH, int stride) {
	//Stride locked to K_WH at the moment

	float max = 0;
	int in_x = threadIdx.x * stride;
	int in_y = threadIdx.y * stride;

	for (size_t ky = 0; ky < K_WH; ky++)
	{
		for (size_t kx = 0; kx < K_WH; kx++)
		{
			float val = input[(in_x + kx) + (in_y + ky) * I_W + blockIdx.x * I_W * I_H];
			if (val > max)
				max = val;
		}
	}

	output[threadIdx.x + threadIdx.y * blockDim.x + blockIdx.x * blockDim.x * blockDim.y] = max;

}

ConvNet::ConvNet(int I_W, int I_H, int nInputs)
{
	LayerData input;
	input.numOutputs = nInputs;
	input.O_W = I_W;
	input.O_H = I_H;

	m_layers.push_back(input);

	m_dataArraySize = input.O_W * input.O_H * input.numOutputs;
}

ConvNet::~ConvNet()
{
	Destroy();
}
/**
	Adds a new layer to the network. This function should not be called after Initialize().
	This is a variatic function and can take any number of parameters of type int. The number of parameters that will give effect depends of the layertype passed is.
	In case no or to few parameters was passed to the specific layertype the new layer will be givven default values to all missing parameters.
	In case to many parameters was passed to the specific layertype all extra parameters vill be ignored.

	@param type the layer type that should be added.
	@param args the number of extra parameters sent to this function, followd by "args" number of parameters. Exemple AddLayer(LAYER_TYPE::ConvLayer, 2, 1, 1) or AddLayer(LAYER_TYPE::ConvLayer, 0)

	@return
*/
void ConvNet::AddLayer(LAYER_TYPE type, int args ...)
{
	va_list args__;
	va_start(args__, args);
	LayerData layerData;

	layerData.type = type;

	int maxInputs;
	if (type == LAYER_TYPE::ConvLayer) {
		maxInputs = 3;

		//Set user values
		for (size_t i = 0; i < args && i < maxInputs; i++)
		{
			int val = va_arg(args__, int);

			switch (i)
			{
			case 0:
				layerData.kernalSize = val;
				break;
			case 1:
				layerData.numOutputs = val;
				break;
			case 2:
				layerData.stride = val;
				break;
			default:
				break;
			}
		}

		//Set default values
		for (size_t i = args; i < maxInputs; i++)
		{
			switch (i)
			{
			case 0:
				layerData.kernalSize = 3;
				break;
			case 1:
				layerData.numOutputs = 1;
				break;
			case 2:
				layerData.numOutputs = 1;
				break;
			default:
				break;
			}
		}


		//Calculate number of kernals needed
		layerData.numKernals = m_layers[m_layers.size() - 1].numOutputs * layerData.numOutputs;
	}
	else if (type == LAYER_TYPE::PoolLayer) {
		maxInputs = 2;

		//Set user values
		for (size_t i = 0; i < args && i < maxInputs; i++)
		{
			int val = va_arg(args__, int);

			switch (i)
			{
			case 0:
				layerData.kernalSize = val;
				break;
			case 1:
				layerData.stride = val;
				break;
			default:
				break;
			}
		}

		//Set default values
		for (size_t i = args; i < maxInputs; i++)
		{
			switch (i)
			{
			case 0:
				layerData.kernalSize = 2;
				break;
			case 1:
				layerData.stride = layerData.kernalSize;
				break;
			default:
				break;
			}
		}

		//Maxpool have same number of outputs as the layer before it
		layerData.numOutputs = m_layers[m_layers.size() - 1].numOutputs;
	}

	//Calculate Layer Output Dimensions
	layerData.O_W = (m_layers[m_layers.size() - 1].O_W - layerData.kernalSize) / layerData.stride + 1;
	layerData.O_H = (m_layers[m_layers.size() - 1].O_H - layerData.kernalSize) / layerData.stride + 1;
	//Calculate number of threads needed. (One thread per output pixel)
	layerData.numThreads = layerData.O_W * layerData.O_H * layerData.numOutputs;

	//Increese amount of datastorage needed on GPU
	m_kernalArraySize += layerData.numKernals * layerData.kernalSize * layerData.kernalSize * sizeof(float);
	m_dataArraySize += layerData.numThreads;

	//Add new layer to the layer vector
	m_layers.push_back(layerData);

	va_end(args__);
}

void ConvNet::Initialize()
{
	Destroy();
	init = true;

	//Cuda Stuff
	hipError_t error;

	error = hipMalloc((void**)&d_dataArray, m_dataArraySize);
	if (error != hipSuccess) {
		std::cout << "Malloc d_dataArray error: " << hipGetErrorString(error) << std::endl;
	}

	error = hipMalloc((void**)&d_kernalArray, m_kernalArraySize);
	if (error != hipSuccess) {
		std::cout << "Malloc d_kernalArray error: " << hipGetErrorString(error) << std::endl;
	}

	//Kernal
	InitializeKernal();
}

void ConvNet::Feed(unsigned char * inputData)
{
	if (!init)
		return;

	//Cuda Stuff
	hipError_t error;

	error = hipMemcpy(d_dataArray, inputData, m_layers[0].O_W * m_layers[0].O_H * m_layers[0].numOutputs, hipMemcpyHostToDevice);
	if (error != hipSuccess) {
		std::cout << "hipMemcpy d_dataArray error: " << hipGetErrorString(error) << std::endl;
	}

	int input_offset = 0;
	int output_offset = 0;
	int kernal_offset = 0;

	dim3 nBlocks;// (m_layers[0].numOutputs);
	dim3 nThreads;// (m_layers[0].O_W, m_layers[0].O_H);

	//ConvLayerFeed<<<nBlocks, nThreads>>>(d_dataArray, m_I_W, m_I_H, m_numInputs, d_dataArray + output_offset, d_kernalArray, 3);
	//error = hipGetLastError();
	//if (error != hipSuccess) {
	//	std::cout << "ConvLayerFeed error: " << hipGetErrorString(error) << std::endl;
	//}

	for (size_t i = 1; i < m_layers.size(); i++)
	{
		input_offset = output_offset;
		output_offset += m_layers[i - 1].numOutputs * m_layers[i - 1].O_W * m_layers[i - 1].O_H;
		kernal_offset += m_layers[i - 1].kernalSize * m_layers[i - 1].kernalSize * m_layers[i - 1].numKernals;

		nBlocks = dim3(m_layers[i].numOutputs);
		nThreads = dim3(m_layers[i].O_W, m_layers[i].O_H);

		if(m_layers[i].type == LAYER_TYPE::ConvLayer)
			ConvLayerFeed << <nBlocks, nThreads >> > (d_dataArray + input_offset, m_layers[i - 1].O_W, m_layers[i - 1].O_H, m_layers[i - 1].numOutputs, d_dataArray + output_offset, d_kernalArray, 3);
		else if (m_layers[i].type == LAYER_TYPE::PoolLayer) {
			MaxPoolLayer << <nBlocks, nThreads >> > (d_dataArray + input_offset, m_layers[i - 1].O_W, m_layers[i - 1].O_H, d_dataArray + output_offset, m_layers[i].kernalSize, m_layers[i].stride);
		}

		error = hipGetLastError();
		if (error != hipSuccess) {
			std::cout << "ConvLayerFeed error #" << i << ": " << hipGetErrorString(error) << std::endl;
		}

	}

}

//void ConvNet::SetKernalData(const void * kernalData, int bytes, int DeviceOffset)
//{
//	hipError_t error;
//
//	error = hipMemcpy(d_kernalArray + DeviceOffset, kernalData, bytes, hipMemcpyHostToDevice);
//	if (error != hipSuccess) {
//		std::cout << "SetKernalData error: " << hipGetErrorString(error) << std::endl;
//	}
//}
//
//void ConvNet::GetKernalData(void * kernalData, int bytes, int DeviceOffset)
//{
//	hipError_t error;
//
//	error = hipMemcpy(kernalData, d_kernalArray + DeviceOffset, bytes, hipMemcpyDeviceToHost);
//	if (error != hipSuccess) {
//		std::cout << "GetKernalData error: " << hipGetErrorString(error) << std::endl;
//	}
//}

void ConvNet::GetData(unsigned char * arrayData, int bytes, int DeviceOffset)
{
	hipError_t error;

	error = hipMemcpy(arrayData, d_dataArray + DeviceOffset, bytes, hipMemcpyDeviceToHost);
	if (error != hipSuccess) {
		std::cout << "GetData error: " << hipGetErrorString(error) << std::endl;
	}
}

void ConvNet::GetData(unsigned char * arrayData, int & dataWidth, int & dataHeight, int maxBytes, int layerIndex, int outputIndex)
{
	hipError_t error;

	dataWidth = m_layers[layerIndex].O_W;
	dataHeight = m_layers[layerIndex].O_H;

	int start = 0;
	int outputSize = m_layers[layerIndex].O_W * m_layers[layerIndex].O_H;
	int read = (outputSize < maxBytes) ? outputSize : maxBytes;

	for (size_t i = 0; i < layerIndex; i++)
	{
		start += m_layers[i].numOutputs * m_layers[i].O_W * m_layers[i].O_H;
	}
	start += outputIndex * m_layers[layerIndex].O_W * m_layers[layerIndex].O_H;

	error = hipMemcpy(arrayData, d_dataArray + start, outputSize, hipMemcpyDeviceToHost);
	if (error != hipSuccess) {
		std::cout << "GetData layer/output error: " << hipGetErrorString(error) << std::endl;
	}
}

void ConvNet::Destroy()
{
	if (!init)
		return;

	//Free Host
	delete[] h_kernalArray;
	//delete[] perLayerData;

	//Free Device
	hipFree(d_dataArray);
	hipFree(d_kernalArray);
}

void ConvNet::InitializeKernal()
{
	int startInitFromIndex = 0;
	h_kernalArray = new float[m_kernalArraySize];

	int left = 0;
	int i = 0;
	int j = 0;
	do
	{
		if (i + 1 < m_layers.size())
			left = m_layers[i].numOutputs * m_layers[i + 1].numOutputs;

		while (left > 0)
		{
			h_kernalArray[j++] = 0;
			h_kernalArray[j++] = -1;
			h_kernalArray[j++] = 0;
			h_kernalArray[j++] = -1;
			h_kernalArray[j++] = 4;
			h_kernalArray[j++] = -1;
			h_kernalArray[j++] = 0;
			h_kernalArray[j++] = -1;
			h_kernalArray[j++] = 0;
			left--;
		}

		i++;
	} while (i < m_layers.size() - 1);

	hipError_t error;

 	error = hipMemcpy(d_kernalArray, h_kernalArray, m_kernalArraySize, hipMemcpyHostToDevice);
	if (error != hipSuccess) {
		std::cout << "SetKernalData error: " << hipGetErrorString(error) << std::endl;
	}
}
